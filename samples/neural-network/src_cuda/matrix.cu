#include "matrix.hh"
#include "nn_exception.hh"

Matrix::Matrix(size_t x_dim, size_t y_dim) :
	shape(x_dim, y_dim), data(nullptr),
	device_allocated(false), host_allocated(false)
{ }

Matrix::Matrix(Shape shape) :
	Matrix(shape.x, shape.y)
{ }



//void Matrix::allocateCudaMemory() {
//	if (!device_allocated) {
//		float* device_memory = nullptr;
//		hipMalloc(&device_memory, shape.x * shape.y * sizeof(float));
//		NNException::throwIfDeviceErrorsOccurred("Cannot allocate CUDA memory for Tensor3D.");
//		data_device = std::shared_ptr<float>(device_memory,
//											 [&](float* ptr){ hipFree(ptr); });
//		device_allocated = true;
//	}
//}
//
//void Matrix::allocateHostMemory() {
//	if (!host_allocated) {
//		data_host = std::shared_ptr<float>(new float[shape.x * shape.y],
//										   [&](float* ptr){ delete[] ptr; });
//		host_allocated = true;
//	}
//}

void Matrix::allocateMemory() {
    float* mem;
    hipMallocManaged(&mem, shape.x*shape.y*sizeof(float));

	data = std::shared_ptr<float>(mem,
		[&](float* ptr){ hipFree(ptr); });
	device_allocated = true;
	host_allocated = true;
	//allocateCudaMemory();
	//allocateHostMemory();
}

void Matrix::allocateMemoryIfNotAllocated(Shape shape) {
	if (!device_allocated && !host_allocated) {
		this->shape = shape;
		allocateMemory();
	}
}

float& Matrix::operator[](const int index) {
	return data.get()[index];
}

const float& Matrix::operator[](const int index) const {
	return data.get()[index];
}
