#include <iostream>
#include <time.h>

#include "fs_initializer.cu.h"
#include "neural_network.hh"
#include "linear_layer.hh"
#include "relu_activation.hh"
#include "sigmoid_activation.hh"
#include "nn_exception.hh"
#include "bce_cost.hh"

#include "coordinates_dataset.hh"
#include <ctime>
#include <cstdio>
#include <iostream>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;


float computeAccuracy(const Matrix& predictions, const Matrix& targets);

void init_device_app()
{
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30);
}

int main() {
	// GPUFS setup 
	int device = 0;
	char* gpudev = getenv("GPUDEVICE");
	if (gpudev != NULL)
		device = atoi(gpudev);

	hipSetDevice(device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	printf("Running on device %d: \"%s\"\n", device, deviceProp.name);
	volatile GPUGlobals* gpuGlobals;
	initializer(&gpuGlobals);
	init_device_app();

	srand( time(NULL) );

	//CoordinatesDataset dataset(100000, 21);
	CoordinatesDataset dataset(10000000, 21);
	BCECost bce_cost;

	NeuralNetwork nn(0.0001, 80000000, 21, gpuGlobals);
	nn.addLayer(new LinearLayer("linear_1", Shape(2, 30), gpuGlobals));
	nn.addLayer(new ReLUActivation("relu_1"));
	nn.addLayer(new LinearLayer("linear_2", Shape(30, 1)));
	nn.addLayer(new SigmoidActivation("sigmoid_output"));

    printf("Beginning training\n");
	// network training
	Matrix Y;

    auto t1 = Clock::now();

	for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) {
        nn.prefetch("input", batch);
    }
	hipError_t error = hipDeviceSynchronize();

	for (int epoch = 0; epoch < 1001; epoch++) {
		float cost = 0.0;

        // Not concerned with actually training, moreso evaluating effect
        // of using gpufs for loading input matrix on runtime performance,
        // so just usin junk
		for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) {
			Y = nn.forward(batch, Shape(10000000, 2)); // dataset.getBatches().at(batch));
			//nn.backprop(Y, dataset.getTargets().at(batch));
			//cost += bce_cost.cost(Y, dataset.getTargets().at(batch));
		}

		if (epoch % 10 == 0) {
			std::cout 	<< "Epoch: " << epoch
						<< ", Cost: " << cost / dataset.getNumOfBatches()
                        << ", Time (s): "
                        << (std::chrono::duration_cast<std::chrono::milliseconds>(Clock::now() - t1).count()) / 1000.0
						<< std::endl;
            t1 = Clock::now();
		}
	}
	return 0;
}

float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
	int m = predictions.shape.x;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i]) {
			correct_predictions++;
		}
	}

	return static_cast<float>(correct_predictions) / m;
}
